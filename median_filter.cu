
#include <hip/hip_runtime.h>
extern "C"{
    __device__ float find_median(float* neighb_array, const int N)
    {
        int i, j;
        float key;

        for (i = 1; i < N; i++)
        {
            key = neighb_array[i];
            j = i - 1;

            while (j >= 0 && neighb_array[j] > key)
            {
                neighb_array[j + 1] = neighb_array[j];
                j = j - 1;
            }
            neighb_array[j + 1] = key;
        }
        return neighb_array[N / 2];
    }
    __global__ void median_filter(float* data_array, const float* padded_array, const int N_IMAGES, const int X, const int Y, const int filter_height, const int filter_width)
    {
        unsigned int id_img = blockIdx.x*blockDim.x + threadIdx.x;
        unsigned int id_x = blockIdx.y*blockDim.y + threadIdx.y;
        unsigned int id_y = blockIdx.z*blockDim.z + threadIdx.z;
        unsigned int n_counter = 0;
        unsigned int img_size =  X * Y;
        unsigned int padded_img_size =  (X + filter_height - 1) * (Y + filter_width - 1);
        unsigned int padded_img_width =  X + filter_height - 1;

        float neighb_array[25];

        if ((id_img < N_IMAGES) && (id_x < X) && (id_y < Y))
        {
            for (int i = id_x; i < id_x + filter_height; i++)
            {
                for (int j = id_y; j < id_y + filter_width; j++)
                {
                    neighb_array[n_counter] = padded_array[(id_img * padded_img_size) + (i * padded_img_width) + j];
                    n_counter += 1;
                }
            }

            if (0)
            {
                find_median(neighb_array, filter_height * filter_width);
                for (int i = 0; i < filter_width * filter_height; i++)
                    printf("%f ", neighb_array[i]);
                printf("\n");
            }

            data_array[(id_img * img_size) + (id_x * X) + id_y] = find_median(neighb_array, filter_height * filter_width);
        }
    }
}