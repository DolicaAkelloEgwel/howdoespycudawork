
#include <hip/hip_runtime.h>
extern "C"{
    __device__ float find_median(float* neighb_array, const int N)
    {
        int i, j;
        float key;

        for (i = 1; i < N; i++)
        {
            key = neighb_array[i];
            j = i - 1;

            while (j >= 0 && neighb_array[j] > key)
            {
                neighb_array[j + 1] = neighb_array[j];
                j = j - 1;
            }
            neighb_array[j + 1] = key;
        }
        return neighb_array[N / 2 + 1];
    }
    __global__ void median_filter(float*** data_array, const float*** padded_array, const int N_IMAGES, const int X, const int Y, const int filter_height, const int filter_width)
    {
        unsigned int id_img = blockIdx.x*blockDim.x + threadIdx.x;
        unsigned int id_x = blockIdx.y*blockDim.y + threadIdx.y;
        unsigned int id_y = blockIdx.z*blockDim.z + threadIdx.z;
        unsigned int n_counter = 0;


        if ((id_img < N_IMAGES) && (id_x < X) && (id_y < Y))
        {
            float neighb_array[20];

            printf("Entering loop\n");
            for (int i = id_x - (filter_height / 2 ); i < id_x + (filter_height / 2); i++)
            {
                for (int j = id_y - (filter_width / 2); j < id_y + (filter_width / 2); j++)
                {
                    neighb_array[n_counter] = padded_array[id_img][id_x][id_y];
                    printf("Array index %d %d %d / idx: %d / idy: %d\n", id_img, i, j, id_x, id_y);
                    n_counter = n_counter + 1;
                }
            }
//            printf("Last value in neighbour array: %lf\n", neighb_array[filter_height * filter_width - 1]);

//            data_array[id_img][id_x][id_y] = find_median(neighb_array, filter_height * filter_width);
        }
    }
}